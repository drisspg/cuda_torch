#include "hip/hip_runtime.h"
#include "include/saturated_cast.h"
#include "utils.h"

#include <ATen/cuda/HIPContext.h>
#include <c10/core/ScalarType.h>
#include <c10/cuda/CUDAException.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/util/Exception.h>

#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>

namespace driss_torch {
using namespace at;

namespace {
__global__ void saturated_cast_kernel_single(
    hip_bfloat16 *input, __hip_fp8_storage_t *output, int n_rows, int n_cols,
    __hip_fp8_interpretation_t out_dtype, hip_bfloat16 *scaler) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // Assume row major
  const int global_index = row * n_cols + col;
  if (row < n_rows && col < n_cols) {
    const hip_bfloat16 scaled_input = __hmul(input[global_index], (*scaler));
    output[global_index] = __hip_cvt_bfloat16raw_to_fp8(
        scaled_input, __hip_saturation_t::__HIP_SATFINITE, out_dtype);
  }
}

__global__ void saturated_cast_kernel_double_coalesced(
    nv_bfloat162 const *__restrict input,
    __hip_fp8x2_storage_t *__restrict output, int n_rows, int n_cols,
    __hip_fp8_interpretation_t out_dtype, hip_bfloat16 const *scaler,
    const int coarse_factor) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = (blockIdx.x * blockDim.x + threadIdx.x) * coarse_factor;
  const int row_stride = n_cols;
  const int col_stride = 1;
  const nv_bfloat162 scale_2 = {(*scaler), (*scaler)};
// Assume row major
#pragma unroll
  for (int i{0}; i < coarse_factor; ++i) {
    col = col + i;
    if (row < n_rows && col < n_cols) {
      const int global_index = row * row_stride + col * col_stride;
      // Need to make a bfloat16x2 from 1 bfloat16
      const nv_bfloat162 scaled_input = __hmul2(input[global_index], scale_2);
      output[global_index] = __hip_cvt_bfloat16raw2_to_fp8x2(
          scaled_input, __hip_saturation_t::__HIP_SATFINITE, out_dtype);
    }
  }
}

__hip_fp8_interpretation_t dtype_map(const ScalarType dtype) {
  switch (dtype) {
  case at::kFloat8_e4m3fn:
    return __hip_fp8_interpretation_t::__HIP_E4M3_FNUZ;
  case at::kFloat8_e5m2:
    return __hip_fp8_interpretation_t::__HIP_E5M2_FNUZ;
  default:
    TORCH_CHECK(false, "Invalid dtype");
  }
}

void dispatch_best_kernel(const Tensor &input, const Tensor &output,
                          __hip_fp8_interpretation_t out_dtype,
                          const Tensor &scale, bool transpose) {
  const int n_rows = input.size(0);
  const int n_cols = input.size(1);
  const int block_size_x = 32;
  const int block_size_y = 32;
  if (n_cols % 2 == 0) {
    // We cast to a 2x8 type, so we need to divide the number of columns by 2
    const auto packed_col_size = n_cols / 2;
    const int coarse_factor = 2;
    const dim3 block(block_size_x, block_size_y);
    const dim3 grid(ceil_div(packed_col_size, block_size_x * coarse_factor),
                    ceil_div(n_rows, block_size_y));
    saturated_cast_kernel_double_coalesced<<<grid, block>>>(
        static_cast<nv_bfloat162 *>(input.data_ptr()),
        static_cast<__hip_fp8x2_storage_t *>(output.data_ptr()), n_rows,
        packed_col_size, out_dtype,
        static_cast<hip_bfloat16 *>(scale.data_ptr()), coarse_factor);
  } else {
    const dim3 block(block_size_x, block_size_y);
    const dim3 grid(ceil_div(n_cols, block_size_x),
                    ceil_div(n_rows, block_size_y));
    saturated_cast_kernel_single<<<grid, block>>>(
        static_cast<hip_bfloat16 *>(input.data_ptr()),
        static_cast<__hip_fp8_storage_t *>(output.data_ptr()), n_rows, n_cols,
        out_dtype, static_cast<hip_bfloat16 *>(scale.data_ptr()));
  }
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}
} // namespace

Tensor saturated_cast(const Tensor &input, ScalarType dtype,
                      const Tensor &scale, bool transpose) {
  TORCH_CHECK(dtype == at::kFloat8_e4m3fn || dtype == at::kFloat8_e5m2,
              "Output tensor must be of type Float8_e4m3fn or Float8_e5m2")
  auto output = torch::empty(input.sizes(), input.options().dtype(dtype));

  TORCH_CHECK(input.scalar_type() == at::kBFloat16,
              "Input tensor must be of type BFloat16");
  TORCH_CHECK(scale.scalar_type() == at::kBFloat16,
              "Scale must be of type BFloat16");
  dispatch_best_kernel(input, output, dtype_map(dtype), scale, transpose);
  return output;
}

} // namespace driss_torch
